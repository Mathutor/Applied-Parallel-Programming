#include "hip/hip_runtime.h"
/* Matrix multiplication: P = M * N.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
  //Multiply the two matrices
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if( col < MATRIX_SIZE && row < MATRIX_SIZE) 
    {
        for(int i = 0; i < MATRIX_SIZE; i++) 
        {
            sum += M.elements[row * MATRIX_SIZE + i] * N.elements[i * MATRIX_SIZE + col];
        }
        P.elements[row * MATRIX_SIZE + col] = sum;
    }

}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
